#include "hip/hip_runtime.h"
#include "../include/wmma_spmm.cuh"
#include "spmm_utils/dense_tile.h"
#include "spmm_utils/sparse_tile.h"
#include "spmm_utils/compute_utils.h"
#include "spmm_utils/output_tile.h"
#include <stdio.h>
#include <mma.h>
//#define ROWNUM 1
#define ROWNUM 4
//#define ROWNUM 2
//#define ROWNUM 2
//#define ROWNUM 16

using namespace nvcuda;

namespace spmm{

//template <typename LoadType, typename IndexType, typename VecType, 
//          typename OutType, typename StoreType, int Tile_N, 
//          int Tile_K, int BlockWidth, int VecLength=8>
//__global__ void wmmaSpmmKernel8(
//    int m, int k, int n, 
//    const int* __restrict__ row_indices, 
//    const int* __restrict__ row_offsets,
//    const int* __restrict__ column_indices,
//    const half* __restrict__ values,
//    const half* __restrict__ rhs_matrix,
//    OutType* __restrict__ output_matrix)
//{
//    // For the wmma based implementation, we have Tile_M = 1
//    int m_index_vec = blockIdx.x;
//    int k_index = blockIdx.y * Tile_K;
//    const int lane_id = threadIdx.x % 4;
//    const int thread_group = threadIdx.x / 4;
//    
//    // Threads that work on different m-dim indices are independent
//    // If we're out of bounds in the m-dimension we can just return
//    if (m_index_vec >= m) return;
//    m_index_vec = __ldg(row_indices + m_index_vec);
//
//    // Load the row offset and calculate the number of nonzeros in the row
//    int row_offset_vec = __ldg(row_offsets + m_index_vec);
//    int nonzeros = __ldg(row_offsets + m_index_vec + 1) - row_offset_vec;
//
//    // For VecLength=8, we don't need the memory aligner
//
//    // Shared memory tiles for the lhs values and indices
//    __shared__ float4 values_tile_array[VecLength * Tile_N];
//    //__shared__ float4 values_tile_array[Tile_N];
//    __shared__ int column_indices_tile_array[Tile_N];
//
//    // Pointers to the shared memory tiles
//    float4 * values_tile = values_tile_array;
//    int* column_indices_tile = column_indices_tile_array;
//
//    // Initialize the pointers to the sparse lhs matrix
//    wmmaSparseTile<LoadType, VecType, VecLength, Tile_N, BlockWidth> sparse_tile_loader(
//        k, row_offset_vec, threadIdx.x, values, column_indices,
//        values_tile, column_indices_tile
//    );
//
//    // Register fragment for the dense matrix values
//    constexpr int kDenseFragmentSize = Tile_N / 4 * 8;
//
//    __align__(16) half dense_matrix_fragment[kDenseFragmentSize];
//
//    // Initialize the pointers to the dense rhs matrix
//    wmmaDenseTile<LoadType, Tile_N, Tile_K, BlockWidth> dense_tile_loader(
//        k, k_index, lane_id, thread_group, rhs_matrix, column_indices_tile, dense_matrix_fragment
//    );
//
//    // Accumulator registers for the output values.
//    constexpr int kOutputFragmentSize = 16;
//    __align__(16) float output_fragment[kOutputFragmentSize] = {};
//    wmmaComputeUtils8<VecType, Tile_N> computer(values_tile, dense_matrix_fragment, output_fragment, lane_id, thread_group);
//
//    //
//    // Begin kernel main loop
//    //
//
//    constexpr int InnerSteps = Tile_N / 4;
//
//    for (; nonzeros >= Tile_N; nonzeros -= Tile_N){
//        sparse_tile_loader.Load();
//        __syncthreads();
//        #pragma unroll
//        for (int n_group_idx = 0; n_group_idx < InnerSteps; n_group_idx ++){
//            dense_tile_loader.LoadRow(n_group_idx);
//        }
//        __threadfence_block();
//        #pragma unroll
//        for (int n_group_idx = 0; n_group_idx < InnerSteps; n_group_idx ++){
//            computer.TileMAC(n_group_idx);
//        }
//        __syncthreads();
//    }
//    asm("");
//
//    sparse_tile_loader.ZeroTiles();
//    __syncthreads();
//    sparse_tile_loader.Residue(nonzeros);
//    __syncthreads();
//    
//    int n_group_idx = 0;
//
//    #pragma unroll
//    for (; n_group_idx < InnerSteps; n_group_idx ++){
//        if (nonzeros < 4) break;
//        dense_tile_loader.LoadRow(n_group_idx);
//        computer.TileMAC(n_group_idx);
//        nonzeros -= 4;
//    }
//    asm("");
//
//    dense_tile_loader.ResidueLoad(n_group_idx, nonzeros);
//    computer.TileMACResidue(n_group_idx);
//
//    wmmaOutputTile8<OutType, StoreType> output_tile_storer(lane_id, thread_group, m_index_vec, 
//        k_index, k, output_fragment, output_matrix);
//    output_tile_storer.Store();
//    
//}
//
//
//template <typename LoadType, typename IndexType, typename VecType, 
//          typename OutType, int Tile_N, 
//          int Tile_K, int BlockWidth, int VecLength=4>
//__global__ void wmmaSpmmKernel4(
//    int m, int k, int n, 
//    const int* __restrict__ row_indices, 
//    const int* __restrict__ row_offsets,
//    const int* __restrict__ column_indices,
//    const half* __restrict__ values,
//    const half* __restrict__ rhs_matrix,
//    OutType* __restrict__ output_matrix)
//{
//    // For the wmma based implementation, we have Tile_M = 1
//    int m_index_vec = blockIdx.x;
//    int k_index = blockIdx.y * Tile_K;
//    const int lane_id = threadIdx.x % 4;
//    const int thread_group = threadIdx.x / 4;
//
//    // Threads that work on different m-dim indices are independent
//    // If we're out of bounds in the m-dimension we can just return
//    if (m_index_vec >= m) return;
//    m_index_vec = __ldg(row_indices + m_index_vec);
//
//    // Load the row offset and calculate the number of nonzeros in the row
//    int row_offset_vec = __ldg(row_offsets + m_index_vec);
//    int nonzeros = __ldg(row_offsets + m_index_vec + 1) - row_offset_vec;
//
//    // Shared memory tiles for the lhs values and indices
//    __shared__ float2 values_tile_array[VecLength * Tile_N];
//    __shared__ int column_indices_tile_array[Tile_N];
//
//    // Pointers to the shared memory tiles
//    float2 * values_tile = values_tile_array;
//    int* column_indices_tile = column_indices_tile_array;
//
//    // Initialize the pointers to the sparse lhs matrix
//    wmmaSparseTile<LoadType, VecType, VecLength, Tile_N, BlockWidth> sparse_tile_loader(
//        k, row_offset_vec, threadIdx.x, values, column_indices,
//        values_tile, column_indices_tile
//    );
//
//    // Register fragment for the dense matrix values
//    constexpr int kDenseFragmentSize = Tile_N / 4 * 8;
//
//    __align__(16) half dense_matrix_fragment[kDenseFragmentSize];
//
//    // Initialize the pointers to the dense rhs matrix
//    wmmaDenseTile<LoadType, Tile_N, Tile_K, BlockWidth> dense_tile_loader(
//        k, k_index, lane_id, thread_group, rhs_matrix, column_indices_tile, dense_matrix_fragment
//    );
//
//
//    // Accumulator registers for the output values.
//    constexpr int kOutputFragmentSize = 8;
//    __align__(16) float output_fragment[kOutputFragmentSize] = {};
//    wmmaComputeUtils4<VecType, Tile_N> computer(values_tile, dense_matrix_fragment, output_fragment, lane_id, thread_group);
//
//    //
//    // Begin kernel main loop
//    //
//
//    constexpr int InnerSteps = Tile_N / 4;
//
//    for (; nonzeros >= Tile_N; nonzeros -= Tile_N){
//        sparse_tile_loader.Load();
//        __syncthreads();
//        #pragma unroll
//        for (int n_group_idx = 0; n_group_idx < InnerSteps; n_group_idx ++){
//            dense_tile_loader.LoadRow(n_group_idx);
//        }
//        __threadfence_block();
//        #pragma unroll
//        for (int n_group_idx = 0; n_group_idx < InnerSteps; n_group_idx ++){
//            computer.TileMAC(n_group_idx);
//        }
//        __syncthreads();
//    }
//    
//    sparse_tile_loader.ZeroTiles();
//    __syncthreads();
//    sparse_tile_loader.Residue(nonzeros);
//    __syncthreads();
//
//    int n_group_idx = 0;
//
//    #pragma unroll
//    for (; n_group_idx < InnerSteps; n_group_idx ++){
//        if (nonzeros < 4) break;
//        dense_tile_loader.LoadRow(n_group_idx);
//        computer.TileMAC(n_group_idx);
//        nonzeros -= 4;
//    }
//    asm("");
//
//    dense_tile_loader.ResidueLoad(n_group_idx, nonzeros);
//    computer.TileMACResidue(n_group_idx);
//
//    wmmaOutputTile4<OutType> output_tile_storer(lane_id, thread_group, m_index_vec, k_index, k, output_fragment, output_matrix);
//    output_tile_storer.Store();
//}

//8-bit integer
template <typename LoadType, typename IndexType, typename VecType, 
          typename OutType, int Tile_N, 
          int Tile_K, int BlockWidth, int VecLength=4>
__global__ void wmmaSpmmKernel4(
    int m_vec, int k, int n, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const int* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    OutType* __restrict__ output_matrix)
{
    // For the wmma based implementation, we have Tile_M = 1
    int m_index_vec = blockIdx.x;
    int k_index = blockIdx.y * Tile_K;
    const int lane_id = threadIdx.x;

    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    if (m_index_vec >= m_vec) return;
    m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;

    // Shared memory tiles for the lhs values and indices
    __shared__ int values_tile_array[Tile_N];
    __shared__ int column_indices_tile_array[Tile_N];

    // each int value has four 8-bit values, padding to avoid bank conflict, assuming Tile_N=32 
    __shared__ int dense_tile_array[Tile_K*Tile_N/4 + 8*7];

    // Pointers to the shared memory tiles
    int* values_tile = values_tile_array;
    int* column_indices_tile = column_indices_tile_array;
    int* dense_tile = dense_tile_array;

    // Initialize the pointers to the sparse lhs matrix
    wmmaSparseTile<LoadType, VecType, VecLength, Tile_N, BlockWidth> sparse_tile_loader(
        k, row_offset_vec, threadIdx.x, values, column_indices,
        values_tile, column_indices_tile
    );

    // Register fragment for the dense matrix values
    //constexpr int kDenseFragmentSize = Tile_N / 4 * 8;
    //__align__(16) half dense_matrix_fragment[kDenseFragmentSize];

    // Initialize the pointers to the dense rhs matrix
    wmmaDenseTile<LoadType, Tile_N, Tile_K, BlockWidth> dense_tile_loader(
        k, k_index/4, lane_id, rhs_matrix, column_indices_tile, dense_tile
    );

    // Accumulator registers for the output values.
    constexpr int kOutputFragmentSize = 16;
    __align__(16) int output_fragment[kOutputFragmentSize] = {};
    wmmaComputeUtils4_8bit<VecType, Tile_N> computer(values_tile, dense_tile, output_fragment, lane_id);

    //
    // Begin kernel main loop
    //

    constexpr int InnerSteps = Tile_N / 16;

    for (; nonzeros >= Tile_N; nonzeros -= Tile_N){
        sparse_tile_loader.Load();
        __syncthreads();
        #pragma unroll
        for (int n_group_idx = 0; n_group_idx < InnerSteps; n_group_idx ++){
            dense_tile_loader.LoadRow(n_group_idx);
            //computer.TileMAC(n_group_idx);
        }
        __threadfence_block();
        #pragma unroll
        for (int n_group_idx = 0; n_group_idx < InnerSteps; n_group_idx ++){
            computer.TileMAC(n_group_idx);
        }
        __syncthreads();
    }
   
    if(nonzeros > 0){
        __syncthreads();
        sparse_tile_loader.Residue(nonzeros);
        __syncthreads();

        int n_group_idx_red = 0;

        #pragma unroll
        for (; n_group_idx_red < InnerSteps; n_group_idx_red++){
            if (nonzeros < 16) break;
            dense_tile_loader.LoadRow(n_group_idx_red);
            computer.TileMAC(n_group_idx_red);
            nonzeros -= 16;
        }
        asm("");

	if(nonzeros > 0){
            dense_tile_loader.ResidueLoad(n_group_idx_red, nonzeros);
            //computer.TileMACResidue(n_group_idx_red);
            computer.TileMAC(n_group_idx_red);
	}
    } 

    wmmaOutputTile4_8bit<OutType> output_tile_storer(lane_id, m_index_vec, k_index, k, output_fragment, output_matrix);
    output_tile_storer.Store();
}

//4-bit 8-v integer larger Tile_K, sm opt
template <typename LoadType, typename IndexType, typename VecType, 
          typename OutType, int Tile_N, 
          int Tile_K, int BlockWidth, int VecLength=8>
__global__ void wmmaSpmmKernel4_4bit_opt(
    int m_vec, int k, int n, int row_num, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const int* __restrict__ row_chunk_offset,
    const int* __restrict__ compact_block_offset,
    const unsigned char* __restrict__ compact_local_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    OutType* __restrict__ output_matrix)
{   
    int m_index_vec = blockIdx.x * row_num;
    int k_index = blockIdx.y * Tile_K;
    m_index_vec += threadIdx.y;
    int local_thread_id = threadIdx.x + threadIdx.y * blockDim.x;
    int local_thread_num = blockDim.y * blockDim.x;
    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    //if (m_index_vec >= m_vec) return;

    int row_offset_vec;
    int nonzeros;

    // TODO: sorting?
    //m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    if (m_index_vec < m_vec){
        row_offset_vec = __ldg(row_offsets + m_index_vec*2);
        nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;
    }else{
        row_offset_vec = 0;
        nonzeros = 0;
    }

    // Shared memory tiles for the lhs values and indices. Seems useless since no reuse. To add more threads on K dim.
    // each int value has eight 4-bit values 
    __shared__ int dense_tile_array[ROWNUM*Tile_N*Tile_K/8];
    __shared__ int dense_row_offsets[ROWNUM*Tile_N];
    __shared__ unsigned char compact_local_indices_sm[ROWNUM*Tile_N];
    unsigned char * compact_local_indices_sm_seg = compact_local_indices_sm + threadIdx.y * 32;
    //__shared__ int dense_tile_array[16384];
    //__shared__ int dense_tile_array[2048];
    //__shared__ int dense_tile_array[1024];
    //__shared__ int dense_tile_array[3072];
    int* dense_tile = dense_tile_array;
    int* offsets_tile = dense_row_offsets;

    int row_block_offset = __ldg(row_chunk_offset + blockIdx.x); 
    int row_block_offset_next = __ldg(row_chunk_offset + blockIdx.x + 1); 

    int col_index_offset_begin = __ldg(compact_block_offset + row_block_offset);

    wmmaDenseTile_4bit_sm_opt<LoadType> dense_tile_loader(
        k/8, k_index/8, local_thread_id, local_thread_num, Tile_K/8, col_index_offset_begin, Tile_N*row_num,
	rhs_matrix, column_indices, compact_block_offset, dense_tile, offsets_tile
    );

    // Accumulator registers for the output values.
    __align__(16) int output_fragment[32] = {};
    wmmaComputeUtils4_4bit_8v_sm_opt<VecType> computer(
        dense_tile, values, compact_local_indices, compact_local_indices_sm_seg, row_offset_vec, threadIdx.x, output_fragment
    );

    int sparse_offset = 0;

    if(row_block_offset < row_block_offset_next){
        dense_tile_loader.LoadRow(row_block_offset);
	//load the dense rhs matrix
        __syncthreads();

        if(sparse_offset < nonzeros){
	//compute tiles
	    if(nonzeros-sparse_offset >= Tile_N){
	        computer.TileMAC(sparse_offset);
	    }
	    else{
	        computer.TileMAC_Residual(sparse_offset);
	    }
            sparse_offset += Tile_N;
	}
    }

    for(int i = row_block_offset+1; i < row_block_offset_next; i++){
        __syncthreads();
        //col_index_offset = __ldg(compact_block_offset + i); 
        //col_index_num = __ldg(compact_block_offset + i + 1) - col_index_offset;
        dense_tile_loader.LoadRow(i);
	//load the dense rhs matrix
        __syncthreads();

        if(sparse_offset < nonzeros){
	//compute tiles
	    if(nonzeros-sparse_offset >= Tile_N){
	        computer.TileMAC(sparse_offset);
	    }
	    else{
	        computer.TileMAC_Residual(sparse_offset);
	    }
            sparse_offset += Tile_N;
	}
    }

    //write back
    if (nonzeros > 0){
        wmmaOutputTile4_4bit_8v_sm_opt<OutType> output_tile_storer(threadIdx.x, VecLength, m_index_vec, k_index, k, output_fragment, output_matrix);
        output_tile_storer.Store();
    }
}

//4-bit 8-v integer larger Tile_K
template <typename LoadType, typename IndexType, typename VecType, 
          typename OutType, int Tile_N, 
          int Tile_K, int BlockWidth, int VecLength=8>
__global__ void wmmaSpmmKernel4_4bit(
    int m_vec, int k, int n, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const int* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    OutType* __restrict__ output_matrix)
{
    // For the wmma based implementation, we have Tile_M = 1
    int m_index_vec = blockIdx.x;
    int k_index = blockIdx.y * Tile_K;
    const int lane_id = threadIdx.x;

    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    if (m_index_vec >= m_vec) return;
    m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;

    // Shared memory tiles for the lhs values and indices.
    __shared__ int values_tile_array[Tile_N];
    __shared__ int column_indices_tile_array[Tile_N];

    // each int value has four 4-bit values, padding to avoid bank conflict, assuming Tile_N=64 
    __shared__ int dense_tile_array[Tile_K*32/8 + 8*3];
    //__shared__ int dense_tile_array[Tile_K*Tile_N/8 + 8*7];
    //__shared__ int dense_tile_array[Tile_K*Tile_N/8];

    // Pointers to the shared memory tiles
    int* values_tile = values_tile_array;
    int* column_indices_tile = column_indices_tile_array;
    int* dense_tile = dense_tile_array;

    // Initialize the pointers to the sparse lhs matrix
    // ToDo: VecType is useless?
    wmmaSparseTile_4bit_8v<LoadType, VecType, VecLength, Tile_N, BlockWidth> sparse_tile_loader(
        k, row_offset_vec, threadIdx.x, values, column_indices,
        values_tile, column_indices_tile
    );

    // Register fragment for the dense matrix values
    //constexpr int kDenseFragmentSize = Tile_N / 4 * 8;
    //__align__(16) half dense_matrix_fragment[kDenseFragmentSize];

    // Initialize the pointers to the dense rhs matrix
    wmmaDenseTile_4bit<LoadType, Tile_N, Tile_K, BlockWidth> dense_tile_loader(
        k, k_index/8, lane_id, rhs_matrix, column_indices_tile, dense_tile
    );

    // Accumulator registers for the output values.
    constexpr int kOutputFragmentSize = 32;
    __align__(16) int output_fragment[kOutputFragmentSize] = {};
    wmmaComputeUtils4_4bit_8v<Tile_N> computer(values_tile, dense_tile, output_fragment, lane_id);

    //
    // Begin kernel main loop
    //

    constexpr int InnerSteps = Tile_N / 32;

    for (; nonzeros >= Tile_N; nonzeros -= Tile_N){
        sparse_tile_loader.Load();
        //__syncthreads();
        #pragma unroll
        for (int n_group_idx = 0; n_group_idx < InnerSteps; n_group_idx ++){
            dense_tile_loader.LoadRow(n_group_idx);
            computer.TileMAC(n_group_idx);
        }
        //__threadfence_block();
        //#pragma unroll
        //for (int n_group_idx = 0; n_group_idx < InnerSteps; n_group_idx ++){
        //    computer.TileMAC(n_group_idx);
        //}
        //__syncthreads();
    }
   
    if(nonzeros > 0){
        //sparse_tile_loader.ZeroTiles();
        //__syncthreads();
        sparse_tile_loader.Residue(nonzeros);
        //__syncthreads();

        int n_group_idx_red = 0;

        #pragma unroll
        for (; n_group_idx_red < InnerSteps; n_group_idx_red++){
            if (nonzeros < 32) break;
            dense_tile_loader.LoadRow(n_group_idx_red);
            computer.TileMAC(n_group_idx_red);
            nonzeros -= 32;
        }
        asm("");

        if(nonzeros > 0){
            dense_tile_loader.ResidueLoad(n_group_idx_red, nonzeros);
            //computer.TileMACResidue(n_group_idx_red);
            computer.TileMAC(n_group_idx_red);
        }
    } 

    wmmaOutputTile4_4bit_8v<OutType> output_tile_storer(lane_id, m_index_vec, k_index, k, output_fragment, output_matrix);
    output_tile_storer.Store();
}

//4-bit integer larger Tile_K
template <typename LoadType, typename IndexType, typename VecType, 
          typename OutType, int Tile_N, 
          int Tile_K, int BlockWidth, int VecLength=4>
__global__ void wmmaSpmmKernel4_4bit(
    int m_vec, int k, int n, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const short* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    OutType* __restrict__ output_matrix)
{
    // For the wmma based implementation, we have Tile_M = 1
    int m_index_vec = blockIdx.x;
    int k_index = blockIdx.y * Tile_K;
    const int lane_id = threadIdx.x;

    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    if (m_index_vec >= m_vec) return;
    m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;

    // Shared memory tiles for the lhs values and indices
    __shared__ int values_tile_array[Tile_N/2];
    __shared__ int column_indices_tile_array[Tile_N];

    // each int value has four 4-bit values, padding to avoid bank conflict, assuming Tile_N=64 
    __shared__ int dense_tile_array[Tile_K*32/8 + 8*3];
    //__shared__ int dense_tile_array[Tile_K*Tile_N/8 + 8*7];
    //__shared__ int dense_tile_array[Tile_K*Tile_N/8];

    // Pointers to the shared memory tiles
    int* values_tile = values_tile_array;
    int* column_indices_tile = column_indices_tile_array;
    int* dense_tile = dense_tile_array;

    // Initialize the pointers to the sparse lhs matrix
    // ToDo: VecType is useless?
    wmmaSparseTile_4bit<LoadType, VecType, VecLength, Tile_N, BlockWidth> sparse_tile_loader(
        k, row_offset_vec, threadIdx.x, values, column_indices,
        values_tile, column_indices_tile
    );

    // Register fragment for the dense matrix values
    //constexpr int kDenseFragmentSize = Tile_N / 4 * 8;
    //__align__(16) half dense_matrix_fragment[kDenseFragmentSize];

    // Initialize the pointers to the dense rhs matrix
    wmmaDenseTile_4bit<LoadType, Tile_N, Tile_K, BlockWidth> dense_tile_loader(
        k, k_index/8, lane_id, rhs_matrix, column_indices_tile, dense_tile
    );

    // Accumulator registers for the output values.
    constexpr int kOutputFragmentSize = 32;
    __align__(16) int output_fragment[kOutputFragmentSize] = {};
    wmmaComputeUtils4_4bit<Tile_N> computer(values_tile, dense_tile, output_fragment, lane_id);

    //
    // Begin kernel main loop
    //

    constexpr int InnerSteps = Tile_N / 32;

    for (; nonzeros >= Tile_N; nonzeros -= Tile_N){
        sparse_tile_loader.Load();
        //__syncthreads();
        #pragma unroll
        for (int n_group_idx = 0; n_group_idx < InnerSteps; n_group_idx ++){
            dense_tile_loader.LoadRow(n_group_idx);
            computer.TileMAC(n_group_idx);
        }
        //__threadfence_block();
        //#pragma unroll
        //for (int n_group_idx = 0; n_group_idx < InnerSteps; n_group_idx ++){
        //    computer.TileMAC(n_group_idx);
        //}
        //__syncthreads();
    }
   
    if(nonzeros > 0){
        //sparse_tile_loader.ZeroTiles();
        //__syncthreads();
        sparse_tile_loader.Residue(nonzeros);
        //__syncthreads();

        int n_group_idx_red = 0;

        #pragma unroll
        for (; n_group_idx_red < InnerSteps; n_group_idx_red++){
            if (nonzeros < 32) break;
            dense_tile_loader.LoadRow(n_group_idx_red);
            computer.TileMAC(n_group_idx_red);
            nonzeros -= 32;
        }
        asm("");

        if(nonzeros > 0){
            dense_tile_loader.ResidueLoad(n_group_idx_red, nonzeros);
            //computer.TileMACResidue(n_group_idx_red);
            computer.TileMAC(n_group_idx_red);
        }
    } 

    wmmaOutputTile4_4bit<OutType> output_tile_storer(lane_id, m_index_vec, k_index, k, output_fragment, output_matrix);
    output_tile_storer.Store();
}

////4-bit integer
//template <typename LoadType, typename IndexType, typename VecType, 
//          typename OutType, int Tile_N, 
//          int Tile_K, int BlockWidth, int VecLength=4>
//__global__ void wmmaSpmmKernel4_4bit(
//    int m_vec, int k, int n, 
//    const int* __restrict__ row_indices, 
//    const int* __restrict__ row_offsets,
//    const int* __restrict__ column_indices,
//    const short* __restrict__ values,
//    const int* __restrict__ rhs_matrix,
//    OutType* __restrict__ output_matrix)
//{
//    // For the wmma based implementation, we have Tile_M = 1
//    int m_index_vec = blockIdx.x;
//    int k_index = blockIdx.y * Tile_K;
//    const int lane_id = threadIdx.x;
//
//    // Threads that work on different m-dim indices are independent
//    // If we're out of bounds in the m-dimension we can just return
//    if (m_index_vec >= m_vec) return;
//    m_index_vec = __ldg(row_indices + m_index_vec);
//
//    // Load the row offset and calculate the number of nonzeros in the row
//    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
//    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;
//
//    // Shared memory tiles for the lhs values and indices
//    __shared__ int values_tile_array[Tile_N/2];
//    __shared__ int column_indices_tile_array[Tile_N];
//
//    // each int value has four 4-bit values, padding to avoid bank conflict, assuming Tile_N=64 
//    __shared__ int dense_tile_array[Tile_K*32/8 + 8*3];
//    //__shared__ int dense_tile_array[Tile_K*Tile_N/8 + 8*7];
//    //__shared__ int dense_tile_array[Tile_K*Tile_N/8];
//
//    // Pointers to the shared memory tiles
//    int* values_tile = values_tile_array;
//    int* column_indices_tile = column_indices_tile_array;
//    int* dense_tile = dense_tile_array;
//
//    // Initialize the pointers to the sparse lhs matrix
//    // ToDo: VecType is useless?
//    wmmaSparseTile_4bit<LoadType, VecType, VecLength, Tile_N, BlockWidth> sparse_tile_loader(
//        k, row_offset_vec, threadIdx.x, values, column_indices,
//        values_tile, column_indices_tile
//    );
//
//    // Register fragment for the dense matrix values
//    //constexpr int kDenseFragmentSize = Tile_N / 4 * 8;
//    //__align__(16) half dense_matrix_fragment[kDenseFragmentSize];
//
//    // Initialize the pointers to the dense rhs matrix
//    wmmaDenseTile_4bit<LoadType, Tile_N, Tile_K, BlockWidth> dense_tile_loader(
//        k, k_index/8, lane_id, rhs_matrix, column_indices_tile, dense_tile
//    );
//
//    // Accumulator registers for the output values.
//    constexpr int kOutputFragmentSize = 16;
//    __align__(16) int output_fragment[kOutputFragmentSize] = {};
//    wmmaComputeUtils4_4bit<Tile_N> computer(values_tile, dense_tile, output_fragment, lane_id);
//
//    //
//    // Begin kernel main loop
//    //
//
//    constexpr int InnerSteps = Tile_N / 32;
//
//    for (; nonzeros >= Tile_N; nonzeros -= Tile_N){
//        sparse_tile_loader.Load();
//        //__syncthreads();
//        #pragma unroll
//        for (int n_group_idx = 0; n_group_idx < InnerSteps; n_group_idx ++){
//            dense_tile_loader.LoadRow(n_group_idx);
//            computer.TileMAC(n_group_idx);
//        }
//        //__threadfence_block();
//        //#pragma unroll
//        //for (int n_group_idx = 0; n_group_idx < InnerSteps; n_group_idx ++){
//        //    computer.TileMAC(n_group_idx);
//        //}
//        //__syncthreads();
//    }
//   
//    if(nonzeros > 0){
//        //sparse_tile_loader.ZeroTiles();
//        //__syncthreads();
//        sparse_tile_loader.Residue(nonzeros);
//        //__syncthreads();
//
//        int n_group_idx_red = 0;
//
//        #pragma unroll
//        for (; n_group_idx_red < InnerSteps; n_group_idx_red++){
//            if (nonzeros < 32) break;
//            dense_tile_loader.LoadRow(n_group_idx_red);
//            computer.TileMAC(n_group_idx_red);
//            nonzeros -= 32;
//        }
//        asm("");
//
//        if(nonzeros > 0){
//            dense_tile_loader.ResidueLoad(n_group_idx_red, nonzeros);
//            //computer.TileMACResidue(n_group_idx_red);
//            computer.TileMAC(n_group_idx_red);
//        }
//    } 
//
//    wmmaOutputTile4_4bit<OutType> output_tile_storer(lane_id, m_index_vec, k_index, k, output_fragment, output_matrix);
//    output_tile_storer.Store();
//}

template <typename IndexType, int Tile_M, int Tile_N, int Tile_K, int BlockWidth>
hipError_t wmmaSpmmEx_4bit(
    int m_vec, int vec_length, int k, int n, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const short* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    dim3 grid_dim(ceil(static_cast<float>(m_vec) / Tile_M), ceil(static_cast<float>(k) / Tile_K), 1);
    dim3 block_dim(BlockWidth, Tile_M, 1);
    switch(vec_length){
        //case 2:
        //    //printf("V=2\n");
        //    wmmaSpmmKernel2<int, int, short, int4, Tile_N, Tile_K, BlockWidth, 2><<<grid_dim, block_dim>>>(
        //        m_vec, k, n, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
        //    break;
        case 4:
            //printf("V=4\n");
            wmmaSpmmKernel4_4bit<int, int, short, int, Tile_N, Tile_K, BlockWidth, 4><<<grid_dim, block_dim>>>(
                m_vec, k, n, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
            break;
        //case 8:
        //    //printf("V=8\n");
        //    wmmaSpmmKernel8<int, int, int2, int4, int2, Tile_N, Tile_K, BlockWidth, 8><<<grid_dim, block_dim>>>(
        //        m_vec, k, n, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
        //    break;
        default:
            printf("Unsupported Vector Length!\n");
    }

    return hipGetLastError();
}

template <typename IndexType, typename VecType, int Tile_M, int Tile_N, int Tile_K, int BlockWidth>
hipError_t wmmaSpmmEx_4bit(
    int m_vec, int vec_length, int k, int n, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    dim3 grid_dim(ceil(static_cast<float>(m_vec) / Tile_M), ceil(static_cast<float>(k) / Tile_K), 1);
    dim3 block_dim(BlockWidth, Tile_M, 1);
    switch(vec_length){
        //case 2:
        //    //printf("V=2\n");
        //    wmmaSpmmKernel2<int, int, short, int4, Tile_N, Tile_K, BlockWidth, 2><<<grid_dim, block_dim>>>(
        //        m_vec, k, n, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
        //    break;
        case 8:
            //printf("V=8\n");
            wmmaSpmmKernel4_4bit<int, int, VecType, int, Tile_N, Tile_K, BlockWidth, 8><<<grid_dim, block_dim>>>(
                m_vec, k, n, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
            break;
        default:
            printf("Unsupported Vector Length!\n");
    }

    return hipGetLastError();
}

template <typename IndexType, typename VecType, int Tile_N, int Tile_K, int BlockWidth>
hipError_t wmmaSpmmEx_4bit_opt(
    int m_vec, int vec_length, int k, int n, int row_num, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const int* __restrict__ row_chunk_offset,
    const int* __restrict__ compact_block_offset,
    const unsigned char* __restrict__ compact_local_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    dim3 grid_dim(ceil(static_cast<float>(m_vec) / row_num), ceil(static_cast<float>(k) / Tile_K), 1);
    dim3 block_dim(BlockWidth, row_num, 1);
    switch(vec_length){
        //case 2:
        //    //printf("V=2\n");
        //    wmmaSpmmKernel2<int, int, short, int4, Tile_N, Tile_K, BlockWidth, 2><<<grid_dim, block_dim>>>(
        //        m_vec, k, n, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
        //    break;
        case 8:
            //printf("V=8\n");
            //printf("Tile_N = %d, row_num = %d \n", Tile_N, row_num);
            //printf("block dim_x %d, block dim_y %d \n", block_dim.x, block_dim.y);
            //printf("grid dim_x %d, grid dim_y %d \n", grid_dim.x, grid_dim.y);
            wmmaSpmmKernel4_4bit_opt<int, int, VecType, int, Tile_N, Tile_K, BlockWidth, 8><<<grid_dim, block_dim>>>(
                m_vec, k, n, row_num, row_indices, row_offsets, column_indices, row_chunk_offset, compact_block_offset, compact_local_indices, values, rhs_matrix, output_matrix);
            break;
        default:
            printf("Unsupported Vector Length!\n");
    }

    return hipGetLastError();
}


template <typename IndexType, int Tile_M, int Tile_N, int Tile_K, int BlockWidth>
hipError_t wmmaSpmmEx_8bit(
    int m_vec, int vec_length, int k, int n, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const int* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    dim3 grid_dim(ceil(static_cast<float>(m_vec) / Tile_M), ceil(static_cast<float>(k) / Tile_K), 1);
    dim3 block_dim(BlockWidth, Tile_M, 1);
    switch(vec_length){
        //case 2:
        //    //printf("V=2\n");
        //    wmmaSpmmKernel2<int, int, short, int4, Tile_N, Tile_K, BlockWidth, 2><<<grid_dim, block_dim>>>(
        //        m_vec, k, n, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
        //    break;
        case 4:
            //printf("V=4\n");
            wmmaSpmmKernel4<int, int, int, int, Tile_N, Tile_K, BlockWidth, 4><<<grid_dim, block_dim>>>(
                m_vec, k, n, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
            break;
        //case 8:
        //    //printf("V=8\n");
        //    wmmaSpmmKernel8<int, int, int2, int4, int2, Tile_N, Tile_K, BlockWidth, 8><<<grid_dim, block_dim>>>(
        //        m_vec, k, n, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
        //    break;
        default:
            printf("Unsupported Vector Length!\n");
    }

    return hipGetLastError();
}

// Function for 8-bit int
hipError_t wmmaSpmm(int m_vec, int vec_length, int k, int n, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const int* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    //printf("8-bit wmmaSpmm\n");
    return wmmaSpmmEx_8bit<int, 1, 32, 64, 32>(m_vec, vec_length, k, n, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
}

// Function for 4-bit int
hipError_t wmmaSpmm(int m_vec, int vec_length, int k, int n, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const short* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    //printf("4-bit wmmaSpmm\n");
    //return wmmaSpmmEx_4bit<int, 1, 64, 64, 32>(m_vec, vec_length, k, n, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
    return wmmaSpmmEx_4bit<int, 1, 64, 128, 32>(m_vec, vec_length, k, n, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
}

// Function for 4-bit int 8v sm opt
hipError_t wmmaSpmm_4b8v_sm_opt(int m_vec, int vec_length, int k, int n, int row_num, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const int* __restrict__ row_chunk_offset,
    const int* __restrict__ compact_block_offset,
    const unsigned char* __restrict__ compact_local_indices,
    const int* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    //printf("4-bit wmmaSpmm\n");
    return wmmaSpmmEx_4bit_opt<int, int, 32, 128, 32>(m_vec, vec_length, k, n, row_num, row_indices, row_offsets, column_indices, row_chunk_offset, compact_block_offset, compact_local_indices, values, rhs_matrix, output_matrix);
}

// Function for 4-bit int 4v sm opt
hipError_t wmmaSpmm_4b8v_sm_opt(int m_vec, int vec_length, int k, int n, int row_num, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const int* __restrict__ row_chunk_offset,
    const int* __restrict__ compact_block_offset,
    const unsigned char* __restrict__ compact_local_indices,
    const short* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    //printf("4-bit wmmaSpmm\n");
    return wmmaSpmmEx_4bit_opt<int, short, 64, 128, 32>(m_vec, vec_length, k, n, row_num, row_indices, row_offsets, column_indices, row_chunk_offset, compact_block_offset, compact_local_indices, values, rhs_matrix, output_matrix);
}

// Function for 4-bit int 8v
hipError_t wmmaSpmm_4b8v(int m_vec, int vec_length, int k, int n, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const int* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    //printf("4-bit wmmaSpmm\n");
    //return wmmaSpmmEx_4bit<int, 1, 64, 64, 32>(m_vec, vec_length, k, n, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
    return wmmaSpmmEx_4bit<int, int, 1, 64, 128, 32>(m_vec, vec_length, k, n, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
}

// Function for 4-bit int 4v
hipError_t wmmaSpmm_4b8v(int m_vec, int vec_length, int k, int n, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const short* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    //printf("4-bit wmmaSpmm\n");
    //return wmmaSpmmEx_4bit<int, 1, 64, 64, 32>(m_vec, vec_length, k, n, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
    return wmmaSpmmEx_4bit<int, short, 1, 64, 128, 32>(m_vec, vec_length, k, n, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
}
//// Function for mixed precision
//hipError_t wmmaSpmm(int m_vec, int vec_length, int k, int n, 
//    const int* __restrict__ row_indices, 
//    const int* __restrict__ row_offsets,
//    const int* __restrict__ column_indices,
//    const half* __restrict__ values,
//    const half* __restrict__ rhs_matrix,
//    float* __restrict__ output_matrix)
//{
//    return wmmaSpmmEx<float4, int, 1, 32, 64, 32>(m_vec, vec_length, k, n, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
//}
//
//// Function for half precision
//hipError_t wmmaSpmm(int m_vec, int vec_length, int k, int n, 
//    const int* __restrict__ row_indices, 
//    const int* __restrict__ row_offsets,
//    const int* __restrict__ column_indices,
//    const half* __restrict__ values,
//    const half* __restrict__ rhs_matrix,
//    half* __restrict__ output_matrix)
//{
//    return wmmaSpmmEx<float4, int, 1, 32, 64, 32>(m_vec, vec_length, k, n, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
//}
//
//// Function for single precision
//hipError_t wmmaSpmm(int m_vec, int vec_length, int k, int n,
//    const int* __restrict__ row_indices,
//    const int* __restrict__ row_offsets,
//    const int* __restrict__ column_indices,
//    const float* __restrict__ values,
//    const float* __restrict__ rhs_matrix,
//    float* __restrict__ output_matrix)
//{
//    printf("wmmaSpmm doesn't support float input.\n");
//    return hipSuccess;
//}

}
